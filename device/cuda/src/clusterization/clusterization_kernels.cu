#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "traccc/cuda/clusterization/clusterization_kernels.cuh"
#include "traccc/clusterization/device/connect_components.hpp"
#include "traccc/clusterization/device/count_cluster_cells.hpp"
#include "traccc/clusterization/device/create_measurements.hpp"
#include "traccc/clusterization/device/find_clusters.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"

namespace traccc::cuda {
namespace kernels {

__global__ void find_clusters(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    /*__shared__ size_t buffer[10000000000];
    cell_container_types::const_device cells_device(cells_view);

    // Get the cells for the current module
    if (threadIdx.x == 0)
    {
        for(int i = 0; i < cells_view.headers.size(); i++)
        {
            const auto& cells = cells_device.at(i).items;
            buffer[i] = cells.size();
        }
    }*/

    device::find_clusters(threadIdx.x + blockIdx.x * blockDim.x, cells_view,
                          sparse_ccl_indices_view, clusters_per_module_view);
}

__global__ void count_cluster_cells(
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view) {

    device::count_cluster_cells(
        threadIdx.x + blockIdx.x * blockDim.x, sparse_ccl_indices_view,
        cluster_prefix_sum_view, cells_prefix_sum_view, cluster_sizes_view);
}

__global__ void connect_components(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    cluster_container_types::view clusters_view) {

    device::connect_components(threadIdx.x + blockIdx.x * blockDim.x,
                               cells_view, sparse_ccl_indices_view,
                               cluster_prefix_sum_view, cells_prefix_sum_view,
                               clusters_view);
}
__global__ void create_measurements(
    const cell_container_types::const_view cells_view,
    cluster_container_types::const_view clusters_view,
    measurement_container_types::view measurements_view) {

    device::create_measurements(threadIdx.x + blockIdx.x * blockDim.x,
                                clusters_view, cells_view, measurements_view);
}

__global__ void form_spacepoints(
    measurement_container_types::const_view measurements_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        measurements_prefix_sum_view,
    spacepoint_container_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, measurements_prefix_sum_view,
                             spacepoints_view);
}
}
}