/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// CUDA Library include(s).
#include "../utils/utils.hpp"
#include "traccc/cuda/utils/definitions.hpp"
#include "traccc/cuda/clusterization/clusterization_cca_algorithm.hpp"
#include "traccc/cuda/clusterization/clusterization_kernels.cuh"

// Project include(s)
#include "traccc/clusterization/device/form_spacepoints.hpp"
#include "traccc/cuda/cca/component_connection.hpp"
#include "traccc/cuda/utils/make_prefix_sum_buff.hpp"
#include "traccc/device/fill_prefix_sum.hpp"
#include "traccc/device/container_d2h_copy_alg.hpp"

// Vecmem include(s).
#include <vecmem/utils/copy.hpp>
#include <vecmem/utils/cuda/copy.hpp>

// System include(s).
#include <algorithm>

// Local include(s)
#include "traccc/cuda/utils/definitions.hpp"

namespace traccc::cuda {

clusterization_cca_algorithm::clusterization_cca_algorithm(
    const traccc::memory_resource& mr, stream& str)
    : m_mr(mr), m_stream(str) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

clusterization_cca_algorithm::output_type clusterization_cca_algorithm::operator()(
    const cell_container_types::host& cells_per_event) const {

    // Vecmem copy object for moving the data between host and device
    vecmem::copy copy;

    // Initialize the device container for cells
    cell_container_types::host cells_device(cells_per_event);

    // Initialize the host container for cells
    //cell_container_types::host cells_host(cells_device);

    // Number of modules
    unsigned int num_modules = cells_per_event.size();
    
    // Work block size for kernel execution
    std::size_t threadsPerBlock = 64;
    
    traccc::cuda::component_connection cc;

    traccc::measurement_container_types::host measurements = cc(cells_per_event);
    
    const auto& measurement_data = get_data(measurements, m_mr.host ? m_mr.host : &(m_mr.main));

    traccc::measurement_container_types::const_view measurements_view(measurement_data);

    std::vector<std::size_t> clusters_per_module_host(num_modules);

    for(int i = 0; i < clusters_per_module_host.size(); i++)
    {
      clusters_per_module_host[i] = measurements_view.items.ptr()[i].size();
    }

    // Create prefix sum buffer
    vecmem::data::vector_buffer meas_prefix_sum_buff = make_prefix_sum_buff(
      m_copy->get_sizes(measurements_view.items), *m_copy, m_mr);

    /*measurement_container_types::buffer measurements_buffer{
      {num_modules, m_mr.main},
      {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
        m_mr.main, m_mr.host}};
    m_copy->setup(measurements_buffer.headers);
    m_copy->setup(measurements_buffer.items);
*/
  //  (*m_copy)(measurements_view.headers, measurements_buffer.headers);
    //(*m_copy)(measurements_view.items, vecmem::get_data(measurements_buffer.items), vecmem::copy::type::host_to_device);

    std::size_t blocksPerGrid = meas_prefix_sum_buff.size()/threadsPerBlock + 1;
    
    spacepoint_container_types::buffer spacepoints_buffer{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
        m_mr.main, m_mr.host}};
    m_copy->setup(spacepoints_buffer.headers);
    m_copy->setup(spacepoints_buffer.items);

    // Invoke spacepoint formation will call form_spacepoints kernel
    traccc::cuda::kernels::form_spacepoints<<<blocksPerGrid, threadsPerBlock>>>(
      measurements_view, meas_prefix_sum_buff, spacepoints_buffer);
    
    // Check for kernel launch errors and Wait for the spacepoint formation
    // kernel to finish
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    return spacepoints_buffer;
  }
}  // namespace traccc::cuda